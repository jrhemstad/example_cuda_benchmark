
#include <benchmark/benchmark.h>

#include <sys/types.h>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/random.h>
#include <cuda/atomic>
#include <synchronization.hpp>

template <typename T>
void BM_weak_sequential_load(benchmark::State &state)
{
  thrust::device_vector<T> v(state.range(0));
  for (auto _ : state) {
    cuda_event_timer raii{state};
    auto const begin = thrust::make_counting_iterator<int32_t>(0);
    auto const end   = thrust::make_counting_iterator<int32_t>(v.size());
    thrust::for_each(
      thrust::device, begin, end, [input_data = v.data().get()] __device__(auto index) {
        volatile auto l = input_data[index];
      });
  }
  state.SetBytesProcessed(int64_t(state.iterations()) * int64_t(state.range(0)) * sizeof(T));
}
BENCHMARK_TEMPLATE(BM_weak_sequential_load, int32_t)
  ->RangeMultiplier(10)
  ->Range(100'000, 1'000'000'000)
  ->UseManualTime()
  ->Unit(benchmark::kMillisecond);

template <typename T>
void BM_weak_random_load(benchmark::State &state)
{
  thrust::device_vector<T> v(state.range(0));

  auto l = [input_size = v.size()] __device__(auto i) {
    thrust::default_random_engine rng;
    thrust::uniform_int_distribution<int32_t> dist(0, input_size);
    rng.discard(i);
    return dist(rng);
  };

  auto const begin = thrust::make_transform_iterator(thrust::make_counting_iterator<int32_t>(0), l);

  auto const end =
    thrust::make_transform_iterator(thrust::make_counting_iterator<int32_t>(v.size()), l);

  for (auto _ : state) {
    cuda_event_timer raii{state};
    thrust::for_each(
      thrust::device, begin, end, [input_data = v.data().get()] __device__(auto index) {
        volatile auto l = input_data[index];
      });
  }
  state.SetBytesProcessed(int64_t(state.iterations()) * int64_t(state.range(0)) * sizeof(T));
}
BENCHMARK_TEMPLATE(BM_weak_random_load, int32_t)
  ->RangeMultiplier(10)
  ->Range(100'000, 1'000'000'000)
  ->UseManualTime()
  ->Unit(benchmark::kMillisecond);

template <typename T>
void BM_atomic_sequential_load(benchmark::State &state)
{
  thrust::device_vector<cuda::atomic<T, cuda::thread_scope_device>> v(state.range(0));
  for (auto _ : state) {
    cuda_event_timer raii{state};
    auto const begin = thrust::make_counting_iterator(0);
    auto const end   = begin + state.range(0);
    thrust::for_each(thrust::device,
                     begin,
                     end,
                     [input_data = v.data().get(), input_size = v.size()] __device__(auto index) {
                       volatile auto l = input_data[index].load(cuda::std::memory_order_relaxed);
                     });
  }
  state.SetBytesProcessed(int64_t(state.iterations()) * int64_t(state.range(0)) * sizeof(T));
}
BENCHMARK_TEMPLATE(BM_atomic_sequential_load, int32_t)
  ->RangeMultiplier(10)
  ->Range(100'000, 1'000'000'000)
  ->UseManualTime()
  ->Unit(benchmark::kMillisecond);

template <typename T>
void BM_atomic_random_load(benchmark::State &state)
{
  thrust::device_vector<cuda::atomic<T, cuda::thread_scope_device>> v(state.range(0));

  auto l = [input_size = v.size()] __device__(auto i) {
    thrust::default_random_engine rng;
    thrust::uniform_int_distribution<int32_t> dist(0, input_size);
    rng.discard(i);
    return dist(rng);
  };

  auto const begin = thrust::make_transform_iterator(thrust::make_counting_iterator<int32_t>(0), l);

  auto const end =
    thrust::make_transform_iterator(thrust::make_counting_iterator<int32_t>(v.size()), l);

  for (auto _ : state) {
    cuda_event_timer raii{state};
    thrust::for_each(
      thrust::device, begin, end, [input_data = v.data().get()] __device__(auto index) {
        volatile auto l = input_data[index].load(cuda::std::memory_order_relaxed);
      });
  }
  state.SetBytesProcessed(int64_t(state.iterations()) * int64_t(state.range(0)) * sizeof(T));
}
BENCHMARK_TEMPLATE(BM_atomic_random_load, int32_t)
  ->RangeMultiplier(10)
  ->Range(100'000, 1'000'000'000)
  ->UseManualTime()
  ->Unit(benchmark::kMillisecond);