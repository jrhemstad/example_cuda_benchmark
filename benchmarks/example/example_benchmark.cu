
#include <benchmark/benchmark.h>

#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <synchronization.hpp>
#include <string>
#include <exception>

inline void throw_cuda_error(hipError_t error, const char *file, unsigned int line)
{
  throw std::runtime_error(std::string{"CUDA error encountered at: " + std::string{file} + ":" +
                                       std::to_string(line) + ": " + std::to_string(error) + " " +
                                       hipGetErrorName(error) + " " + hipGetErrorString(error)});
}

/**
 * @brief Error checking macro for CUDA runtime API functions.
 *
 * Invokes a CUDA runtime API function call, if the call does not return
 * hipSuccess, invokes hipGetLastError() to clear the error and throws an
 * exception detailing the CUDA error that occurred
 */
#define CUDA_TRY(call)                                            \
  do {                                                            \
    hipError_t const status = (call);                            \
    if (hipSuccess != status) {                                  \
      hipGetLastError();                                         \
      throw_cuda_error(status, __FILE__, __LINE__); \
    }                                                             \
  } while (0)


void BM_cuda_async_no_threshold(benchmark::State &state)
{
  hipStream_t s;
  hipStreamCreate(&s);

  auto constexpr block_size = 256;
  auto const grid_size      = (state.range(0) + block_size - 1) / block_size;

  for (auto _ : state) {
    int *ptr;
    CUDA_TRY(hipMallocAsync(&ptr, state.range(0) * sizeof(int), s));
    CUDA_TRY(hipFreeAsync(ptr, s));
    CUDA_TRY(hipStreamSynchronize(s));
  }
  hipStreamDestroy(s);
}
BENCHMARK(BM_cuda_async_no_threshold)
  ->RangeMultiplier(10)
  ->Range(100'000, 100'000'000)
  ->Unit(benchmark::kMicrosecond);

void BM_cuda_async_threshold(benchmark::State &state)
{
  hipStream_t s;
  CUDA_TRY(hipStreamCreate(&s));

  auto constexpr block_size = 256;
  auto const grid_size      = (state.range(0) + block_size - 1) / block_size;

  hipMemPool_t mempool;
  CUDA_TRY(hipDeviceGetDefaultMemPool(&mempool, 0));
  uint64_t threshold = UINT64_MAX;
  CUDA_TRY(hipMemPoolSetAttribute(mempool, hipMemPoolAttrReleaseThreshold, &threshold));

  for (auto _ : state) {
    int *ptr;
    CUDA_TRY(hipMallocAsync(&ptr, state.range(0) * sizeof(int), s));
    CUDA_TRY(hipFreeAsync(ptr, s));
    CUDA_TRY(hipStreamSynchronize(s));
  }
  CUDA_TRY(hipStreamDestroy(s));
}
BENCHMARK(BM_cuda_async_threshold)
  ->RangeMultiplier(10)
  ->Range(100'000, 100'000'000)
  ->Unit(benchmark::kMicrosecond);
